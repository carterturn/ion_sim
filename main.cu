#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include <cmath>
#include <cstring>
#include <unistd.h>

#include <GLFW/glfw3.h>
#include <GL/gl.h>

#include "particle.h"
#include "input_loader.h"

using namespace std;

#define BLOCK_SIZE 32

#define EPSILON 0.000001;

__device__ __host__ bool is_zero(double value){
	return abs(value) < EPSILON;
}

__device__ __host__ double sign(double value){
	return value >= 0 ? 1.0 : -1.0;
}

__device__ __host__ double half_abs(double value){
	return value >= 0 ? value : 0.0;
}

__global__ void calculate_force_matrix(particle * particles, double * forces_x, double * forces_y,
				       simulation_config * config){
	int ind = (blockIdx.x * BLOCK_SIZE) + threadIdx.x;

	if(ind >= config->number_particles * (config->number_particles + 1) / 2){
		return;
	}

	int i = ind / config->number_particles;
	int j = ind - i * config->number_particles;

	if(j < i){
		ind = config->number_particles * config->number_particles - ind + config->number_particles - 1;
		i = config->number_particles - i;
		j = config->number_particles - j - 1;
	}

	double d_x = (particles[i].x - particles[j].x) * config->meters_per_square;
	double d_y = (particles[i].y - particles[j].y) * config->meters_per_square;
	double F = (K * particles[i].q * particles[j].q) /
		(d_x*d_x + d_y*d_y);
	double F_y;
	double F_x;
	double hyp = sqrt(d_x*d_x + d_y*d_y);
	F_y = is_zero(d_y) ? 0.0 : F * d_y / hyp;
	F_x = is_zero(d_x) ? 0.0 : F * d_x / hyp;

	forces_x[ind] = F_x;
	forces_y[ind] = F_y;
	forces_x[j * config->number_particles + i] = -1.0 * F_x;
	forces_y[j * config->number_particles + i] = -1.0 * F_y;
}

__global__ void move_particles(particle * particles, double * forces_x, double * forces_y,
			       simulation_config * config){
	int i = (blockIdx.x * BLOCK_SIZE) + threadIdx.x;

	if(i >= config->number_particles){
		return;
	}

	for(int j = 0; j < config->number_particles; j++){
		if(j > i){
			int ind = (i + 1) * (2*config->number_particles - i) / 2 + (j - i - 1) -
				config->number_particles;
			particles[i].vx += forces_x[ind] * config->dt / particles[i].m;
			particles[i].vy += forces_y[ind] * config->dt / particles[i].m;
		}
		else if(i > j){
			int ind = (j + 1) * (2*config->number_particles - j) / 2 + (i - j - 1) -
				config->number_particles;
			particles[i].vx += -1.0 * forces_x[ind] * config->dt / particles[i].m;
			particles[i].vy += -1.0 * forces_y[ind] * config->dt / particles[i].m;
		}
	}
	
	particles[i].x += particles[i].vx * config->dt / config->meters_per_square;
	if(particles[i].x > config->width){
		particles[i].x = config->width;
		particles[i].vx = -particles[i].vx * config->wall_elasticity;
	}
	if(particles[i].x < 0){
		particles[i].x = 0;
		particles[i].vx = -particles[i].vx * config->wall_elasticity;
	}
	particles[i].y += particles[i].vy * config->dt / config->meters_per_square;
	if(particles[i].y > config->height){
		particles[i].y = config->height;
		particles[i].vy = -particles[i].vy * config->wall_elasticity;
	}
	if(particles[i].y < 0){
		particles[i].y = 0;
		particles[i].vy = -particles[i].vy * config->wall_elasticity;
	}
}

int main(int argc, char* argv[]){

	if(argc < 2){
		cout << "Usage: ./a.out [config file]\n";
		return -1;
	}

	simulation_config config = load_config(argv[1]);

	glfwInit();
	GLFWwindow * window = glfwCreateWindow(config.width, config.height, "Ion Simulator", NULL, NULL);

	if(!window){
		return -1;
	}

	glfwMakeContextCurrent(window);
	glOrtho(0, config.width, 0, config.height, -1.0, 1.0);

	vector<particle> cpu_particles = load_particles(config.particles_file);
	config.number_particles = cpu_particles.size();

	int move_blocks = config.number_particles / BLOCK_SIZE;
	if(config.number_particles % BLOCK_SIZE != 0){
		move_blocks++;
	}

	int particle_data_size = config.number_particles * sizeof(particle);

	particle * gpu_particles = NULL;
	hipError_t error = hipMalloc(&gpu_particles, particle_data_size);
	if(error != hipSuccess){
		cout << hipGetErrorString(error) << "\n";
	}
	error = hipMemcpy(gpu_particles, &cpu_particles.front(), particle_data_size, hipMemcpyHostToDevice);
	if(error != hipSuccess){
		cout << hipGetErrorString(error) << "\n";
	}

	simulation_config * gpu_config = NULL;
	error = hipMalloc(&gpu_config, sizeof(simulation_config));
	if(error != hipSuccess){
		cout << hipGetErrorString(error) << "\n";
	}
	error = hipMemcpy(gpu_config, &config, sizeof(simulation_config), hipMemcpyHostToDevice);
	if(error != hipSuccess){
		cout << hipGetErrorString(error) << "\n";
	}

	double * forces_x = NULL;
	double * forces_y = NULL;
	int force_matrix_size = config.number_particles * config.number_particles;
	int force_matrix_memory = force_matrix_size * sizeof(double);
	error = hipMalloc(&forces_x, force_matrix_memory);
	if(error != hipSuccess){
		cout << hipGetErrorString(error) << "\n";
	}
	error = hipMalloc(&forces_y, force_matrix_memory);
	if(error != hipSuccess){
		cout << hipGetErrorString(error) << "\n";
	}
	int force_blocks = force_matrix_size / BLOCK_SIZE;
	if(force_matrix_size % BLOCK_SIZE != 0){
		force_blocks++;
	}

	pair_index * force_indicies = NULL;
	error = hipMalloc(&force_indicies, force_matrix_size * sizeof(pair_index));
	if(error != hipSuccess){
		cout << hipGetErrorString(error) << "\n";
	}
	pair_index * cpu_force_indicies = new pair_index[force_matrix_size];

	for(int i = 0; i < force_matrix_size; i++){
		cpu_force_indicies[i].i = 0;
		int step = config.number_particles - 1;
		for(int j = 0; j < force_matrix_size; step--){
			j += step;
			if(i < j){
				cpu_force_indicies[i].j = i - j + step + cpu_force_indicies[i].i + 1;
				break;
			}
			cpu_force_indicies[i].i++;
		}
	}
	
	error = hipMemcpy(force_indicies, cpu_force_indicies, force_matrix_size * sizeof(pair_index),
			   hipMemcpyHostToDevice);
	if(error != hipSuccess){
		cout << hipGetErrorString(error) << "\n";
	}

	int tick_count = 0;
	for(double t = 0.0; t < config.total_time; t += config.dt){
		hipMemset(forces_x, 0, force_matrix_memory);
		hipMemset(forces_y, 0, force_matrix_memory);
		
		calculate_force_matrix<<<force_blocks, BLOCK_SIZE>>>(gpu_particles, forces_x, forces_y, gpu_config,
			force_indicies);
		
		move_particles<<<move_blocks, BLOCK_SIZE>>>(gpu_particles, forces_x, forces_y, gpu_config);

		error = hipMemcpy(&cpu_particles.front(), gpu_particles,
				   particle_data_size, hipMemcpyDeviceToHost);
		if(error != hipSuccess){
			cout << hipGetErrorString(error) << "\n";
		}

		if(tick_count % config.ticks_per_display == 0){
		
			glClear(GL_COLOR_BUFFER_BIT);
			glBegin(GL_POINTS);
			glColor3f(0.0f, 1.0f, 0.0f);
			for(int i = 0; i < config.number_particles; i++){
				glVertex2i((int) cpu_particles[i].x, (int) cpu_particles[i].y);
			}
			glEnd();
			glfwSwapBuffers(window);
			cout << t << "\n";
		}
		tick_count++;
	}

	hipFree(forces_x);
	hipFree(forces_y);
	hipFree(gpu_particles);
	hipFree(gpu_config);

	return 0;
}
